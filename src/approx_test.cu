#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  Prashant Pandey <ppandey@cs.stonybrook.edu>
 *                  Rob Johnson <robj@vmware.com>
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <openssl/rand.h>
#include <chrono>
#include<iostream>

#include "include/gqf_int.cuh"
#include "include/gqf_file.cuh"
#include "hashutil.cuh"
#include "include/gqf.cuh"
//#include "src/gqf.cu"

#define CYCLES_PER_SECOND 1601000000

#define MAX_VALUE(nbits) ((1ULL << (nbits)) - 1)
#define BITMASK(nbits)((nbits) == 64 ? 0xffffffffffffffff : MAX_VALUE(nbits))

int main(int argc, char** argv) {
	if (argc < 2) {
		fprintf(stderr, "Please specify the first and second indices to test CQF.\n");
		exit(1);

	}
	QF* qf;

	auto setup_start =  std::chrono::high_resolution_clock::now();

	printf("Start of everything.\n");
	uint64_t qbits = atoi(argv[1]);
	uint64_t rbits = 8;
	uint64_t vbits = 8;
	uint64_t nhashbits = qbits + rbits;
	uint64_t nslots = (1ULL << qbits);
	// //this can be changed to change the % it fills up
	// uint64_t nvals = 95 * nslots / 100;
	// //uint64_t nvals =  nslots/2;
	// //uint64_t nvals = 4;
	// //uint64_t nvals = 1;
	// uint64_t key_count = 1;
	// uint64_t* vals;

	qf_malloc_device(&qf, qbits);


	uint64_t nvals = .5 * (1ULL << qbits);

	uint64_t * vals;


	// /* Initialise the CQF */
	// if (!qf_malloc(&qf, nslots, nhashbits, 0, QF_HASH_INVERTIBLE, false, 0)) {
	// 	fprintf(stderr, "Can't allocate CQF.\n");
	// 	abort();
	// }



	// /*
	// if (!qf_initfile(&qf, nslots, nhashbits, 0, QF_HASH_NONE, 0,
	// 								 "/tmp/mycqf.file")) {
	// 	fprintf(stderr, "Can't allocate CQF.\n");
	// 	abort();
	// }
	// */
	// qf_set_auto_resize(&qf, false);
	// /* Generate random values */
	vals = (uint64_t*)malloc(nvals * sizeof(uint64_t));
	RAND_bytes((unsigned char*)vals, sizeof(*vals) * nvals);
	// //uint64_t* _vals;
	for (uint64_t i = 0; i < nvals; i++) {
		//nslots is the range - why are these different?
		vals[i] = (1 * vals[i]) % (1ULL << qbits);
	 	vals[i] = hash_64(vals[i], BITMASK(nhashbits));
	}


	uint8_t * first;
	uint8_t * second;

	first = (uint8_t * ) malloc(nvals*sizeof(uint8_t));
	second = (uint8_t *) malloc(nvals*sizeof(uint8_t));

	RAND_bytes((unsigned char*)first, sizeof(*first) * nvals);
	RAND_bytes((unsigned char*)second, sizeof(*second) * nvals);

	for (uint64_t i=0; i < nvals; i++){


		//0-4
		first[i] = first[i] % 5;

		//5-9
		second[i] = second[i] % 5 + 5;

	}

	//copy over

	uint64_t * dev_hashes;
	uint8_t * dev_firsts;
	uint8_t * dev_seconds;

	hipMalloc((void ** )&dev_hashes, nvals*sizeof(uint64_t));
	hipMalloc((void ** )&dev_firsts, nvals*sizeof(uint8_t));
	hipMalloc((void ** )&dev_seconds, nvals*sizeof(uint8_t));


	hipMemcpy(dev_hashes, vals, nvals*sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_firsts, first, nvals*sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_seconds, second, nvals*sizeof(uint8_t), hipMemcpyHostToDevice);



	uint64_t * counter1;
	uint64_t * counter2;

	hipMallocManaged((void **)&counter1, sizeof(uint64_t));
	hipMallocManaged((void **)&counter2, sizeof(uint64_t));

	counter1[0] = 0;
	counter2[0] = 0;


	uint64_t * max;
	uint64_t * min;
	uint64_t * total;


	hipMallocManaged((void **)&max, sizeof(uint64_t));
	hipMallocManaged((void **)&min, sizeof(uint64_t));
	hipMallocManaged((void **)&total, sizeof(uint64_t));

	max[0] = 0;
	min[0] = 0;
	total[0] = 0;;
	// // vals = (uint64_t *) malloc(nvals * sizeof(uint64_t));
	// // for (uint64_t i =0l; i< nvals; i++){
	// // 	vals[i] = i;
	// // }

	// srand(0);
	// /* Insert keys in the CQF */
	// printf("starting kernel\n");
	// qf_gpu_launch(&qf, vals, nvals, key_count, nhashbits, nslots);
	// hipDeviceSynchronize();

	// printf("GPU launch succeeded\n");
	// fflush(stdout);


	//remove slots per lock

	hipDeviceSynchronize();

	auto setup_end =  std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> setup_diff = setup_end-setup_start;

	std::cout << "Setup done in " << setup_diff.count() << " seconds\n";


	hipDeviceSynchronize();

	auto start =  std::chrono::high_resolution_clock::now();



	insert_multi_kmer_kernel<<<(nvals-1)/32 +1, 32>>>(qf, dev_hashes, dev_firsts, dev_seconds, nvals, counter1, max, min, total);
	insert_multi_kmer_kernel<<<(nvals-1)/32 +1, 32>>>(qf, dev_hashes, dev_firsts, dev_seconds, nvals, counter2, max, min, total);
    
	
	hipDeviceSynchronize();

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Sans buffers, Inserted " << nvals << " in " << diff.count() << " seconds\n";

 	printf("Inserts per second: %f\n", nvals/diff.count());

 	printf("Inserts per find: %f\n", 2*nvals/diff.count());

 	printf("Positive rate for first round: %llu/%llu: %f\n", counter1[0], nvals, 1.0*counter1[0]/nvals);
 	printf("Positive rate for second round: %llu/%llu: %f\n", counter2[0], nvals, 1.0*counter2[0]/nvals);


 	uint64_t found_nslots = host_qf_get_nslots(qf);
	uint64_t occupied = host_qf_get_num_occupied_slots(qf);

 	printf("Fill ratio: %f %llu %llu\n", 1.0*occupied/found_nslots, occupied, found_nslots);

	printf("Min time: %f %llu/%llu\n", 1.0*min[0]/CYCLES_PER_SECOND, min[0], CYCLES_PER_SECOND);
	
	printf("Max time: %f %llu/%llu\n", 1.0*max[0]/CYCLES_PER_SECOND, max[0], CYCLES_PER_SECOND);

	printf("Average time: %f %llu/%llu\n", 1.0*total[0]/(nvals*CYCLES_PER_SECOND), total[0], nvals*CYCLES_PER_SECOND);


 	hipFree(counter1);
 	hipFree(counter2);

 	hipFree(max);
 	hipFree(min);
 	hipFree(total);

 	hipFree(dev_hashes);
 	hipFree(dev_firsts);
 	hipFree(dev_seconds);

 	qf_destroy_device(qf);


	return 0;

}
